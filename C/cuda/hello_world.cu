#include<iostream>
#include<hip/hip_runtime.h>

using std::cout;
using std::endl;


__global__ void kernel(void) {

}

void check_gpu_info() {
    int dev = 0;
    hipDeviceProp_t devProp;
    if (hipGetDeviceProperties(&devProp, dev)) {
        std::cout << "get properties fail" << std::endl;
        return;
    }
    cout << "device " << dev << ": " << devProp.name << endl;
    cout << "num of SM: " << devProp.multiProcessorCount << endl;
    cout << "shared memory of each block: " 
        << devProp.sharedMemPerBlock
        << endl;
    cout << "max thread num of each block: "
        << devProp.maxThreadsPerBlock
        << endl;
    cout << "max thread num of each SM: "
        << devProp.maxThreadsPerMultiProcessor
        << endl;
    cout << "max thread wrap num of each SM: "
        << devProp.maxThreadsPerMultiProcessor / 32
        << endl;

}

int main(void) {
    kernel<<<2, 2>>>();
    printf("hello world\n");
    check_gpu_info();
    return 0;
}
